#include "hip/hip_runtime.h"
#include "main.h"

void spectr(int size, vector<double>& data_channel, hipfftComplex* data_Host, hipfftComplex* data_dev, vector<double>& power)
{
    hipfftHandle plan;  // ������� ���������� cuFFT

    // �������� ������
    for (int i = 0; i < size; i++)
    {
        data_Host[i].x = data_channel[i];
        data_Host[i].y = (double)0;
    }

    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);

    hipMemset(data_dev, 0, sizeof(hipfftComplex) * size);  // ������������� ��������� 0
    hipMemcpy(data_dev, data_Host, size * sizeof(hipfftComplex), hipMemcpyHostToDevice);  // �������� �� ������ ����� � ������ ����������

    hipfftExecC2C(plan, data_dev, data_dev, HIPFFT_FORWARD);  // ��������� cuFFT, ������������� ��������������
    hipMemcpy(data_Host, data_dev, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);  // �������� �� ������ ���������� � ������ �����

    for (int i = 0; i < size; i++)
        power[i] = sqrt(data_Host[i].x * data_Host[i].x + data_Host[i].y * data_Host[i].y);  // ��������������� ������ � ��������

    hipfftDestroy(plan);  // ���������� ����������
}

void spectrogram_from_signal_cuda(wav_header_t& header, int samples_count, vector<double>& data_channel_1, vector<double>& data_channel_2)
{
    int size = 0;  // ������ ������ �������� ���������� �������

    if (header.numChannels == 1) size = samples_count;
    else size = samples_count / 2;

    vector<double> power_ch1(size);  // ������ � ������� ������
    vector<double> power_ch2(size);  // ������ �� ������� ������
    vector<double> frequency;  // �������� �� x

    for (int i = 0; i < size; i++)
        frequency.push_back(((double)(header.sampleRate) / (double)(size)) * i);

    hipfftComplex* data_dev; // ������ �� ������� ����������
    hipfftComplex* data_Host = (hipfftComplex*)malloc(size * sizeof(hipfftComplex));  // ������ �� ������� �����

    hipMalloc((void**)&data_dev, sizeof(hipfftComplex) * size);  // �������� ������ �� ����������

    spectr(size, data_channel_1, data_Host, data_dev, power_ch1);
    spectr(size, data_channel_2, data_Host, data_dev, power_ch2);

    viewGraph(header, frequency, power_ch1, power_ch2, "Spectrogram", 2);

    hipFree(data_dev); // ���������� �����
    //hipFree(data_Host); // ���������� �����
    free(data_Host);  // ���������� �����
}