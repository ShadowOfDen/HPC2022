#include "hip/hip_runtime.h"
#include "main.h"

#define BLOCK_SIZE 1024

__global__ void cufftMultiply(hipfftComplex* idata, hipfftComplex* odata, int size)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID < size)
    {
        odata[threadID].x = sqrt(idata[threadID].x * idata[threadID].x + idata[threadID].y * idata[threadID].y);  // ��������������� ������ � ��������
    }
}


void spectr(int size, vector<double>& data_channel, hipfftComplex* data_Host, hipfftComplex* data_dev, vector<double>& power)
{
    hipfftHandle plan;  // ������� ���������� cuFFT

    // �������� ������
    for (int i = 0; i < size; i++)
    {
        data_Host[i].x = data_channel[i];
        data_Host[i].y = (double)0;
    }

    dim3 dimBlock(BLOCK_SIZE); // ���� ������
    dim3 dimGrid((size + BLOCK_SIZE - 1) / dimBlock.x); // ����� �������

    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);

    hipMemset(data_dev, 0, sizeof(hipfftComplex) * size);  // ������������� ��������� 0
    hipMemcpy(data_dev, data_Host, size * sizeof(hipfftComplex), hipMemcpyHostToDevice);  // �������� �� ������ ����� � ������ ����������

    hipfftExecC2C(plan, data_dev, data_dev, HIPFFT_FORWARD);  // ��������� cuFFT, ������������� ��������������
    cufftMultiply << <dimGrid, dimBlock >> > (data_dev, data_dev, size);
    hipMemcpy(data_Host, data_dev, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);  // �������� �� ������ ���������� � ������ �����

    for (int i = 0; i < size; i++)
        //power[i] = sqrt(data_Host[i].x * data_Host[i].x + data_Host[i].y * data_Host[i].y);  // ��������������� ������ � ��������
        power[i] = data_Host[i].x;  // ��������������� ������ � ��������

    hipfftDestroy(plan);  // ���������� ����������
}

void spectrogram_from_signal_cuda(wav_header_t& header, int samples_count, vector<double>& data_channel_1, vector<double>& data_channel_2)
{
    int size = 0;  // ������ ������ �������� ���������� �������

    if (header.numChannels == 1) size = samples_count;
    else size = samples_count / 2;

    vector<double> power_ch1(size);  // ������ � ������� ������
    vector<double> power_ch2(size);  // ������ �� ������� ������
    vector<double> frequency;  // �������� �� x

    for (int i = 0; i < size; i++)
        frequency.push_back(((double)(header.sampleRate) / (double)(size)) * i);

    hipfftComplex* data_dev; // ������ �� ������� ����������
    hipfftComplex* data_Host = (hipfftComplex*)malloc(size * sizeof(hipfftComplex));  // ������ �� ������� �����

    hipMalloc((void**)&data_dev, sizeof(hipfftComplex) * size);  // �������� ������ �� ����������

    spectr(size, data_channel_1, data_Host, data_dev, power_ch1);
    spectr(size, data_channel_2, data_Host, data_dev, power_ch2);

    viewGraph(header, frequency, power_ch1, power_ch2, "Spectrogram", 2);

    hipFree(data_dev); // ���������� �����
    //hipFree(data_Host); // ���������� �����
    free(data_Host);  // ���������� �����
}