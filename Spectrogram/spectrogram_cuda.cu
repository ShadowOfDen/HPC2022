#include "hip/hip_runtime.h"
#include "main.h"

#define BLOCK_SIZE 1024

__global__ void cufftMultiply(hipfftComplex* idata, hipfftComplex* odata, int size)
{
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID < size)
    {
        odata[threadID].x = sqrt(idata[threadID].x * idata[threadID].x + idata[threadID].y * idata[threadID].y);  // Преобразовываем данные в мощность
    }
}


void spectr(int size, vector<double>& data_channel, hipfftComplex* data_Host, hipfftComplex* data_dev, vector<double>& power)
{
    hipfftHandle plan;  // Создаем дескриптор cuFFT

    // Исходные данные
    for (int i = 0; i < size; i++)
    {
        data_Host[i].x = data_channel[i];
        data_Host[i].y = (double)0;
    }

    dim3 dimBlock(BLOCK_SIZE); // блок потока
    dim3 dimGrid((size + BLOCK_SIZE - 1) / dimBlock.x); // сетка потоков

    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);

    hipMemset(data_dev, 0, sizeof(hipfftComplex) * size);  // Первоначально заполняем 0
    hipMemcpy(data_dev, data_Host, size * sizeof(hipfftComplex), hipMemcpyHostToDevice);  // Копируем из памяти хоста в память устройства

    hipfftExecC2C(plan, data_dev, data_dev, HIPFFT_FORWARD);  // Выполняем cuFFT, положительное преобразование
    cufftMultiply << <dimGrid, dimBlock >> > (data_dev, data_dev, size);
    hipMemcpy(data_Host, data_dev, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);  // Копируем из памяти устройства в память хоста

    for (int i = 0; i < size; i++)
        //power[i] = sqrt(data_Host[i].x * data_Host[i].x + data_Host[i].y * data_Host[i].y);  // Преобразовываем данные в мощность
        power[i] = data_Host[i].x;  // Преобразовываем данные в мощность

    hipfftDestroy(plan);  // Уничтожаем дескриптор
}

void spectrogram_from_signal_cuda(wav_header_t& header, int samples_count, vector<double>& data_channel_1, vector<double>& data_channel_2)
{
    int size = 0;  // Задаем размер согласно количеству каналов

    if (header.numChannels == 1) size = samples_count;
    else size = samples_count / 2;

    vector<double> power_ch1(size);  // Данные с первого канала
    vector<double> power_ch2(size);  // Данные со второго канала
    vector<double> frequency;  // Значения по x

    for (int i = 0; i < size; i++)
        frequency.push_back(((double)(header.sampleRate) / (double)(size)) * i);

    hipfftComplex* data_dev; // Данные на стороне устройства
    hipfftComplex* data_Host = (hipfftComplex*)malloc(size * sizeof(hipfftComplex));  // Данные на стороне хоста

    hipMalloc((void**)&data_dev, sizeof(hipfftComplex) * size);  // Выделяем память на устройстве

    spectr(size, data_channel_1, data_Host, data_dev, power_ch1);
    spectr(size, data_channel_2, data_Host, data_dev, power_ch2);

    viewGraph(header, frequency, power_ch1, power_ch2, "Spectrogram", 2);

    hipFree(data_dev); // освободить место
    //hipFree(data_Host); // освободить место
    free(data_Host);  // освободить место
}
